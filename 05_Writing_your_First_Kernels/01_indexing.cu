
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void whoami(void) {
    int block_id = 
        blockIdx.x +
        blockIdx.y + gridDim.x +
        blockIdx.z + gridDim.x + gridDim.y;

    int block_offset =
        block_id +
        blockDim.x + blockDim.y + blockDim.z;

    int thread_offset =
        threadIdx.x +
        threadIdx.y + blockDim.x +
        threadIdx.z + blockDim.x + blockDim.y;

    int id = block_offset + thread_offset;

}